#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

//THIS IS THE TREE REDUCTION CODE
#define TILE_WIDTH 16
__constant__ float Mask[10000];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    int shared_width = TILE_WIDTH + K - 1;
    extern __shared__ float sharedMemory[];
    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) Mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define sharedMemTree(i2, i1, i0) sharedMemory[i2 * TILE_WIDTH * Channel + i1 * Channel + i0]
    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int b_out = bx;
    int m_out = by;
    int c_in = tz;


    int Width_grid = (Width_out - 1) / TILE_WIDTH + 1;
    int w = TILE_WIDTH * (bz % Width_grid) + tx;
    int h = TILE_WIDTH * (bz / Width_grid) + ty;

    int heightBoundary = TILE_WIDTH * (bz / Width_grid);
    int widthBoundary = TILE_WIDTH * (bz % Width_grid);

    if (h < Height_out && w < Width_out) {
        float result = 0;
        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++)
            {
                result += in_4d(b_out, c_in, h + p, w + q) * mask_4d(m_out, c_in, p, q);
            }
        }
        sharedMemTree(ty, tx, c_in) = result;

        for (int stride = 1; stride < Channel; stride *= 2) {
            __syncthreads();
            if ((tz % (2 * stride) == 0) && (tz + stride < Channel)) {
                sharedMemTree(ty, tx, tz) += sharedMemTree(ty, tx, tz + stride);
            }
        }
        __syncthreads();
        if (tz == 0) {
            out_4d(b_out, m_out, h, w) = sharedMemTree(ty, tx, 0);
        }
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
    #undef sharedMemTree
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipMalloc((void**)device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Mask), host_mask, (Map_out * Channel * K * K) * sizeof(float));
    //hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, Channel);
    dim3 gridDim(Batch, Map_out, ceil((float)(Height - K + 1) / TILE_WIDTH) * ceil((float)(Width - K + 1) / TILE_WIDTH));
    conv_forward_kernel<<<gridDim, blockDim, Channel * TILE_WIDTH * TILE_WIDTH * sizeof(float)>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    hipMemcpy(host_output, device_output, Batch * Map_out * Height_out * Width_out * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_output);
    hipFree(device_input);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}




